#include "hip/hip_runtime.h"
%%writefile max_with_reduction_warp_shuffle.cu
// !nvcc max_with_reduction_warp_shuffle.cu -o max_with_reduction_warp_shuffle
// !./max_with_reduction_warp_shuffle
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <float.h>

//#define N 104857600
//#define N 1048576
//#define N 1047576
//#define N 1024

#define N 671078400 // The input data size to reduce
#define GRID_SIZE 65535 // Number of blocks in the grid
#define THREADS_PER_BLOCK 1024 // 1024 threads per block

__global__ void reduce_max(float *gdata, float *out) {
    // __shared__ sdata: Each block of threads has its own shared memory space, and this memory is accessible only by the threads within the same block.
    // Shared memory to store partial sums from each warp
    __shared__ float sdata[32];


    int tid = threadIdx.x; // Local thread ID with in a block
    int idx = threadIdx.x + blockDim.x * blockIdx.x; // Global thread index across the grid
    float val = -FLT_MAX; // To store the max value, init with Lowest possible number, thread is important here
    unsigned mask = 0xFFFFFFFFU; // Mask used for warp shuffle operations
    int lane = threadIdx.x % warpSize; // Lane ID within a warp (0 to warpSize-1)
    int warpID = threadIdx.x / warpSize; // Warp ID within a block

    // Grid-stride loop: each thread processes multiple elements spaced by the grid size
    while (idx < N) {
        val = max(gdata[idx], val); // Get the max value
        idx += gridDim.x * blockDim.x; // Move to the next element this thread is responsible for
    }

    // 1st warp-shuffle reduction
    // Perform warp-level reduction using shuffle operations
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        // Shuffle down and find max across the warp
        val = max(__shfl_down_sync(mask, val, offset), val);
    }

    // Store the result of each warp's reduction in shared memory
    if (lane == 0) sdata[warpID] = val;
    // Synchronize threads within the block to ensure all warps have written their results
    __syncthreads(); // put warp results in shared mem

    // hereafter, just warp 0
    // Further reduction within the first warp to combine the warp results
    if (warpID == 0) {
        // Load warp's result or zero if outside
        val = (tid < blockDim.x / warpSize) ? sdata[lane] : -FLT_MAX;
        // final warp-shuffle reduction
        // Final shuffle down and add within the first warp
        for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            val = max(__shfl_down_sync(mask, val, offset), val);
        }

        // First thread of the block updates the global sum atomically
        if (tid == 0) {
            // Use atomicMax with integer representation of float
            int* addr_as_int = (int*)out; // Cast the address of out to int pointer
            int old = *addr_as_int, assumed;
            do {
                assumed = old;
                old = atomicMax(addr_as_int, __float_as_int(max(val, __int_as_float(assumed))));
            } while (assumed != old);
        }
    }
}

int main() {

    // Host and device data pointers and output variable
    float *h_data, *d_data, *d_out, h_out = -FLT_MAX;
    size_t bytes = N * sizeof(float);

    // Allocate memory on the host
    h_data = (float*)malloc(bytes);

    // Initialize host array
    for(int i = 0; i < N; i++) {
        h_data[i] = (float)i; // Generating numbers
    }

    // Allocate memory on the device
    hipMalloc(&d_data, bytes);
    hipMalloc(&d_out, sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_data, h_data, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_out, &h_out, sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block sizes
//    int blockSize = THREADS_PER_BLOCK;
//    int gridSize = (N + blockSize - 1) / blockSize;

    // Launch the kernel with a fixed grid and block size
    reduce_max<<<GRID_SIZE, THREADS_PER_BLOCK>>>(d_data, d_out);

    // Copy result back to host
    hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    printf("Max Value: %f\n", h_out);

    // Cleanup, free host and device memory
    free(h_data);
    hipFree(d_data);
    hipFree(d_out);

    return 0;
}